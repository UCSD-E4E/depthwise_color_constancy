#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define IN_TILE_WIDTH 32   
#define KERNEL_SIZE 1      
#define OUT_TILE_WIDTH (IN_TILE_WIDTH - 2 * KERNEL_SIZE)  

__global__
void softmaxDepthAdverging(
  float * ds,
  float * output,
  float * depth,
  int width, 
  int height, 
  int channels
)
{
  //Prep for tiling both depth and ds to preprocess some data
  // Tiled Convolution implmenetation based on 
  // Programming Massively Parallel Processors
  __shared__ float ds_tile[IN_TILE_WIDTH][IN_TILE_WIDTH][3];
  __shared__ float depth_tile[IN_TILE_WIDTH][IN_TILE_WIDTH];

  int col = blockIdx.x * OUT_TILE_WIDTH + threadIdx.x - KERNEL_SIZE;
  int row = blockIdx.y * OUT_TILE_WIDTH + threadIdx.y - KERNEL_SIZE;
  int pixel_idx = (row * width + col) * channels;
  if (row>=0 && row < height && col >= 0 && col < width) {
    depth_tile[threadIdx.y][threadIdx.x] = depth[row * width + col];
    ds_tile[threadIdx.y][threadIdx.x][0] = ds[pixel_idx];
    ds_tile[threadIdx.y][threadIdx.x][1] = ds[pixel_idx + 1];
    ds_tile[threadIdx.y][threadIdx.x][2] = ds[pixel_idx + 2];
  } else {
    depth_tile[threadIdx.y][threadIdx.x] = 0; //incase of access, this goes to 0
    ds_tile[threadIdx.y][threadIdx.x][0] = 1;
    ds_tile[threadIdx.y][threadIdx.x][1] = 0;
    ds_tile[threadIdx.y][threadIdx.x][2] = 0;
  }
  
  __syncthreads();

  int x = threadIdx.x;
  int y = threadIdx.y;
  float eps = 10;  
  float exp_sum = 0;  
  
  
  if (row>=0 && row < height && col >= 0 && col < width) {
    if(x>= KERNEL_SIZE && x<=OUT_TILE_WIDTH + KERNEL_SIZE && y>= KERNEL_SIZE && y<=OUT_TILE_WIDTH + KERNEL_SIZE) {
      //TODO: Find better defintion for softmax
      // compute denom for softmax                       
      for (int i = -KERNEL_SIZE; i <= KERNEL_SIZE; i++) { // TODO generalize this
        for (int j = -KERNEL_SIZE; j <= KERNEL_SIZE; j++) { //like with a footprint sys
          int nx = x + j;
          int ny = y + i;
          if (ny>=0 && ny < height && nx >= 0 && nx < width) {
            if (nx >= 0 && nx < IN_TILE_WIDTH && ny >= 0 && ny < IN_TILE_WIDTH) {                              
                exp_sum += expf(-abs(depth_tile[ny][nx] - depth_tile[y][x]) - eps); 
            }  
          }             
        }
      } 
      // so now $\text{texp_sum} = \sum_{n \in N} e^{-|n_d - t_d|}}
      // softmax of a neighbor n is $\frac{e^{-|n_d - t_d|}}{exp_sum}$ 

      // Apply avging to each color channels
      for (int c = 0; c < channels; c++) {  
        float avg_color = 0;                  
        
        //convolution here
        for (int i = -KERNEL_SIZE; i <= KERNEL_SIZE; i++) {
          for (int j = -KERNEL_SIZE; j <= KERNEL_SIZE; j++) {
            int nx = x + j;
            int ny = y + i;    
            
            // $\sum_{n \in N} softmax(n) * color(n)
            if (ny>=0 && ny < height && nx >= 0 && nx < width) {
              if (nx >= 0 && nx < IN_TILE_WIDTH && ny >= 0 && ny < IN_TILE_WIDTH) {                      
                //weight of the softmax by the color
                float softmax_weight =  expf(-abs(depth_tile[ny][nx] - depth_tile[y][x]) - eps)/exp_sum;
                avg_color += softmax_weight *  ds_tile[ny][nx][c]; 
              }        
            }    
          }
        }
        // set the color to the updated color
        output[pixel_idx + c] = avg_color;                                              
      } 
    }
  }                       
} 


float* load_bin_files(const char* path, size_t element_size, size_t num_elements) {
  FILE *file = fopen(path, "rb");
  if (!file) {
    perror("Failed to open file");
  }

  float *h_temp = (float*)malloc(num_elements * element_size);
  fread(h_temp, element_size, num_elements, file);
  fclose(file);

  return h_temp;
}

float * depthwiseColorConsistency(
  int iterations,
  int image_width,
  int image_height,
  int image_num_channels,
  float alpha,
  float *  h_depth_map_ptr,
  float *  h_image_ptr
) 
{
  float *d_depth_map_ptr, *d_in_image_ptr, *d_temp_ptr, *d_a_c_ptr;
  float beta = 1.f - alpha;
  int num_pixels =  image_width * image_height * image_num_channels;
  int depth_size =  image_width * image_height;

  //depth map of image
  hipMalloc(&d_depth_map_ptr, depth_size * sizeof(float));
  hipMemcpy(d_depth_map_ptr, h_depth_map_ptr, depth_size * sizeof(float), hipMemcpyHostToDevice);
  
  //original image, needs to be kept to keep a_c stable each iteration
  hipMalloc(&d_in_image_ptr, num_pixels * sizeof(float));
  hipMemcpy(d_in_image_ptr, h_image_ptr, num_pixels * sizeof(float), hipMemcpyHostToDevice);

  //a_c is the avged color image each iteration, starts with original image
  hipMalloc(&d_a_c_ptr, num_pixels * sizeof(float));
  hipMemcpy(d_a_c_ptr, h_image_ptr, num_pixels * sizeof(float), hipMemcpyHostToDevice);
  
  //temp image for holding raw illumiant map
  hipMalloc(&d_temp_ptr, num_pixels * sizeof(float));

  // +10 is a workaround for this missing a column
  // TODO fix this workaround
  dim3 dimGrid(
    ceil((image_width + IN_TILE_WIDTH)/IN_TILE_WIDTH) + 10, 
    ceil((image_height + IN_TILE_WIDTH)/IN_TILE_WIDTH) + 10
  );
  dim3 dimBlock(IN_TILE_WIDTH, IN_TILE_WIDTH);

  //cublas handlers
  hipblasHandle_t handle; 
  hipblasCreate(&handle);

  // Conduct Depthwise Operation
  for (size_t i = 0; i < iterations; i++) { 
    softmaxDepthAdverging<<<dimGrid, dimBlock>>>(
      d_a_c_ptr, d_temp_ptr, d_depth_map_ptr, 
      image_width, image_height, image_num_channels);
    
      //https://stackoverflow.com/questions/56043539/cublassgemm-row-major-multiplication#:~:text=As%20you%20said%2C%20cuBLAS%20interprets,for%20the%20column%2Dmajor%20interpretation.
    //According to here, we can just do the tranpose instead. I'm fine with that. 
    hipblasSgeam(
      handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
      image_num_channels, image_height * image_width,
      &alpha, d_temp_ptr, image_num_channels,
      &beta, d_in_image_ptr, image_num_channels,
      d_a_c_ptr, image_num_channels
    );
  }

  hipDeviceSynchronize();
  
  //write the output for the new lim to test out!
  float *h_out = (float*)malloc(num_pixels * sizeof(float));
  hipMemcpy(h_out, d_a_c_ptr, num_pixels * sizeof(float), hipMemcpyDeviceToHost);

  // Free memory
  hipblasDestroy(handle);
  hipFree(d_in_image_ptr);
  hipFree(d_depth_map_ptr);
  hipFree(d_temp_ptr);
  hipFree(d_a_c_ptr);
  return h_out;
}

//for python freeing
void free_array(float* arr) {
  free(arr);
}

//for python freeing
void free_array(double* arr) {
  free(arr);
}

int main(void)
{
  // image parameters
 //number of pixels * number of channels

  int iterations = 1000;
  float alpha = 0.9999f;

  int width = 640; //480,640
  int height = 480;
  int channels = 3; //Ideally RGB for now
  int num_pixels =  width * height * channels;
  char * image_path = "data/realsense_tests/living_room_0046b_out_1-color.bin";
  char * depth_path = "data/realsense_tests/living_room_0046b_out_1-depth.bin";
  char * output_path = "data/realsense_tests/living_room_0046b_out_1-lim.bin";
  

  // Init Memory
  float *depth, *ds;
  ds = load_bin_files(image_path, sizeof(float), num_pixels);
  depth = load_bin_files(depth_path, sizeof(float), width * height);
  
  float* h_out = depthwiseColorConsistency(
    iterations,
    width,
    height,
    channels,
    alpha,
    depth,
    ds
  );

  FILE* out_f = fopen(output_path, "wb");
  fwrite(h_out, sizeof(float), num_pixels, out_f);
  fclose(out_f);
  
  free(h_out);
  free(ds);
  free(depth);
  
  return 0;
}