#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <iostream>

#define IN_TILE_WIDTH 32
#define KERNEL_SIZE 1
#define OUT_TILE_WIDTH (IN_TILE_WIDTH - 2 * KERNEL_SIZE)

__global__ void softmaxDepthAdverging(
    float *ds,
    float *output,
    float *depth,
    int width,
    int height,
    int channels)
{
    // Prep for tiling both depth and ds to preprocess some data
    //  Tiled Convolution implmenetation based on
    //  Programming Massively Parallel Processors
    __shared__ float ds_tile[IN_TILE_WIDTH][IN_TILE_WIDTH][3];
    __shared__ float depth_tile[IN_TILE_WIDTH][IN_TILE_WIDTH];

    int col = blockIdx.x * OUT_TILE_WIDTH + threadIdx.x - KERNEL_SIZE;
    int row = blockIdx.y * OUT_TILE_WIDTH + threadIdx.y - KERNEL_SIZE;
    int pixel_idx = (row * width + col) * channels;
    if (row >= 0 && row < height && col >= 0 && col < width)
    {
        depth_tile[threadIdx.y][threadIdx.x] = depth[row * width + col];
        ds_tile[threadIdx.y][threadIdx.x][0] = ds[pixel_idx];
        ds_tile[threadIdx.y][threadIdx.x][1] = ds[pixel_idx + 1];
        ds_tile[threadIdx.y][threadIdx.x][2] = ds[pixel_idx + 2];
    }
    else
    {
        depth_tile[threadIdx.y][threadIdx.x] = 0; // incase of access, this goes to 0
        ds_tile[threadIdx.y][threadIdx.x][0] = 1;
        ds_tile[threadIdx.y][threadIdx.x][1] = 0;
        ds_tile[threadIdx.y][threadIdx.x][2] = 0;
    }

    __syncthreads();

    int x = threadIdx.x;
    int y = threadIdx.y;
    float eps = 10;
    float exp_sum = 0;

    if (row >= 0 && row < height && col >= 0 && col < width)
    {
        if (x >= KERNEL_SIZE && x <= OUT_TILE_WIDTH + KERNEL_SIZE && y >= KERNEL_SIZE && y <= OUT_TILE_WIDTH + KERNEL_SIZE)
        {
            //  compute denom for softmax
            for (int i = -KERNEL_SIZE; i <= KERNEL_SIZE; i++)
            { // TODO generalize this
                for (int j = -KERNEL_SIZE; j <= KERNEL_SIZE; j++)
                { // like with a footprint sys
                    int nx = x + j;
                    int ny = y + i;
                    if (ny >= 0 && ny < height && nx >= 0 && nx < width)
                    {
                        if (nx >= 0 && nx < IN_TILE_WIDTH && ny >= 0 && ny < IN_TILE_WIDTH)
                        {
                            exp_sum += expf(-abs(depth_tile[ny][nx] - depth_tile[y][x]) - eps);
                        }
                    }
                }
            }
            // so now $\text{texp_sum} = \sum_{n \in N} e^{-|n_d - t_d|}}
            // softmax of a neighbor n is $\frac{e^{-|n_d - t_d|}}{exp_sum}$

            // Apply avging to each color channels
            for (int c = 0; c < channels; c++)
            {
                float avg_color = 0;

                // convolution here
                for (int i = -KERNEL_SIZE; i <= KERNEL_SIZE; i++)
                {
                    for (int j = -KERNEL_SIZE; j <= KERNEL_SIZE; j++)
                    {
                        int nx = x + j;
                        int ny = y + i;

                        // $\sum_{n \in N} softmax(n) * color(n)
                        if (ny >= 0 && ny < height && nx >= 0 && nx < width)
                        {
                            if (nx >= 0 && nx < IN_TILE_WIDTH && ny >= 0 && ny < IN_TILE_WIDTH)
                            {
                                // weight of the softmax by the color
                                float softmax_weight = expf(-abs(depth_tile[ny][nx] - depth_tile[y][x]) - eps) / exp_sum;
                                avg_color += softmax_weight * ds_tile[ny][nx][c];
                            }
                        }
                    }
                }
                // set the color to the updated color
                output[pixel_idx + c] = avg_color;
            }
        }
    }
}

extern "C" void depthwiseColorConsistency(unsigned int iterations, unsigned int image_width,
                                          unsigned int image_height, unsigned int image_num_channels,
                                          float alpha, const float *h_depth_map_ptr,
                                          const float *h_image_ptr, float *h_out)
{
    float *d_depth_map_ptr, *d_in_image_ptr, *d_temp_ptr, *d_a_c_ptr;
    float beta = 1.f - alpha;
    int num_pixels = image_width * image_height * image_num_channels;
    int depth_size = image_width * image_height;

    // depth map of image
    hipMalloc(&d_depth_map_ptr, depth_size * sizeof(float));
    hipMemcpy(d_depth_map_ptr, h_depth_map_ptr, depth_size * sizeof(float),
               hipMemcpyHostToDevice);

    // original image, needs to be kept to keep a_c stable each iteration
    hipMalloc(&d_in_image_ptr, num_pixels * sizeof(float));
    hipMemcpy(d_in_image_ptr, h_image_ptr, num_pixels * sizeof(float),
               hipMemcpyHostToDevice);

    // a_c is the avged color image each iteration, starts with original image
    hipMalloc(&d_a_c_ptr, num_pixels * sizeof(float));
    hipMemcpy(d_a_c_ptr, h_image_ptr, num_pixels * sizeof(float),
               hipMemcpyHostToDevice);

    // temp image for holding raw illumiant map
    hipMalloc(&d_temp_ptr, num_pixels * sizeof(float));

    // +10 is a workaround for this missing a column
    // TODO fix this workaround
    dim3 dimGrid(ceil((image_width + IN_TILE_WIDTH) / IN_TILE_WIDTH) + 10,
                 ceil((image_height + IN_TILE_WIDTH) / IN_TILE_WIDTH) + 10);
    dim3 dimBlock(IN_TILE_WIDTH, IN_TILE_WIDTH);

    // cublas handlers
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Conduct Depthwise Operation
    for (unsigned int i = 0; i < iterations; i++)
    {
        softmaxDepthAdverging<<<dimGrid, dimBlock>>>(
            d_a_c_ptr, d_temp_ptr, d_depth_map_ptr, image_width, image_height,
            image_num_channels);

        // https://stackoverflow.com/questions/56043539/cublassgemm-row-major-multiplication
        // According to here, we can just do the tranpose instead. I'm fine with
        // that.
        hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, image_num_channels,
                    image_height * image_width, &alpha, d_temp_ptr,
                    image_num_channels, &beta, d_in_image_ptr, image_num_channels,
                    d_a_c_ptr, image_num_channels);
    }

    hipDeviceSynchronize();

    // write the output for the new lim to test out!
    // We expect the output to be the same size as the input.
    hipMemcpy(h_out, d_a_c_ptr, num_pixels * sizeof(float),
               hipMemcpyDeviceToHost);

    // Free memory
    hipblasDestroy(handle);
    hipFree(d_in_image_ptr);
    hipFree(d_depth_map_ptr);
    hipFree(d_temp_ptr);
    hipFree(d_a_c_ptr);
}
